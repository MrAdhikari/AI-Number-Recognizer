#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "kernel.cuh"



#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

int Comp_Minimap(E_Layer & devEL, int i);
void calc_dw_db(E_Layer * EL, E_Layer & devEL, float* ds, float* devds, float* dw, float real_cost, int arrsize);
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

//__global__ void addKernel(int *c, const int *a, const int *b)
//{
//    int i = threadIdx.x;
//    c[i] = a[i] + b[i];
//}

//how it works
//__global__ void dot_w_X_f(float* refw, float* w, float* X)
//{
//    if (blockIdx.x < 20 && threadIdx.x < 784)
//    {
//        refw[blockIdx.x * 784 + threadIdx.x] = (w[blockIdx.x * 784 + threadIdx.x] * X[threadIdx.x]);
//
//    }
//}
// example 
//__global__ void Comp_y(float* Y, float* refw, float* b)
//{
//    if (threadIdx.x < 20 && blockIdx.x < 1)
//    {
//        for (int i = 0; i < 784; i++)
//        {
//            Y[threadIdx.x] += refw[threadIdx.x * 784 + i];
//        }
//        Y[threadIdx.x] += b[threadIdx.x];
//        Y[threadIdx.x] = (1.0f / (1.0f + powf(2.71828f, -(Y[threadIdx.x]))));
//    }
//
//}

// 'bl' is block and 'th' is thread
__global__ void Sqs_X(float* X)
{
    if (X[threadIdx.x] > 1)
    {
        X[threadIdx.x] = 1;
    }
    else if (X[threadIdx.x] < 0)
    {
        X[threadIdx.x] = 0;
    }
    /*if (threadIdx.x < 784)
    {
        X[threadIdx.x] = (1.0f / (1.0f + powf(2.71828f, -(X[threadIdx.x]))));

    }*/
}
__global__ void Dot_w_X(float *refw, float *w,float *X, int w_row, int w_col)
{
    if (blockIdx.x < w_row && threadIdx.x < w_col)
    {
        refw[blockIdx.x * w_col + threadIdx.x] = (w[blockIdx.x * w_col + threadIdx.x] * X[threadIdx.x]);

    }
}

// 'bl' is block and 'th' is thread
__global__ void Comp_y(float* Y,float* refw,float* b, int w_row, int w_col)
{
    if (threadIdx.x < w_row && blockIdx.x < 1)
    {
        Y[threadIdx.x] = 0;
        for (int i = 0; i < w_col; i++)
        {
            Y[threadIdx.x] += refw[threadIdx.x * w_col + i];
        }
        Y[threadIdx.x] += b[threadIdx.x];


        //use of ReLU
        if (Y[threadIdx.x] > 1)
        {
            Y[threadIdx.x] = 1;
        }
        else if(Y[threadIdx.x] < 0)
        {
            Y[threadIdx.x] = 0;
        }
        //use of Sigmoid
        //Y[threadIdx.x] = (1.0f / (1.0f + powf(2.71828f, -(Y[threadIdx.x]))));
    }
    
}

int c_main(E_Layer *EL)
{
    E_Layer devEL;

    float real_cost = 0;
    float calc_cost = 0;


    /*-----memeory allocation ---------*/
    hipMalloc((void**)&devEL.LF.X, 784 * 5 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devEL.LF.w, 784 * 20 * sizeof(float));
    hipMalloc((void**)&devEL.LF.refw, 784 * 20 * sizeof(float));
    hipMalloc((void**)&devEL.LF.b, 20 * sizeof(float));
    hipMalloc((void**)&devEL.LF.Y, 20 * sizeof(float));

    // devHF[0] mem allocation
    //hipMalloc((void**)&devLF.X, 784 * sizeof(float));
    hipMalloc((void**)&devEL.LH[0].w, 20 * 20 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devEL.LH[0].refw, 20 * 20 * sizeof(float));
    hipMalloc((void**)&devEL.LH[0].b, 20 * sizeof(float));
    hipMalloc((void**)&devEL.LH[0].Y, 20 * sizeof(float));

    // devHF[0] mem allocation
    //hipMalloc((void**)&devLF.X, 784 * sizeof(float));
    hipMalloc((void**)&devEL.LH[1].w, 20 * 20 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devEL.LH[1].refw, 20 * 20 * sizeof(float));
    hipMalloc((void**)&devEL.LH[1].b, 20 * sizeof(float));
    hipMalloc((void**)&devEL.LH[1].Y, 20 * sizeof(float));

    //devLL mem allocation
    hipMalloc((void**)&devEL.LL.w, 20 * 10 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devEL.LL.refw, 20 * 10  * sizeof(float));
    hipMalloc((void**)&devEL.LL.b, 10 * sizeof(float));
    hipMalloc((void**)&devEL.LL.Y, 10 * sizeof(float));

    hipMemcpy(devEL.LF.X, EL->io.X, 784 *5 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed from io failed");

    hipMemcpy(devEL.LF.w, EL->LF.w, 784 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");
    hipMemcpy(devEL.LF.b, EL->LF.b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");
    hipMemcpy(devEL.LF.Y, EL->LF.Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");

    hipMemcpy(devEL.LH[0].w, EL->LH[0].w, 20 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devEL.LH[0].b, EL->LH[0].b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devEL.LH[0].Y, EL->LH[0].Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");

    hipMemcpy(devEL.LH[1].w, EL->LH[1].w, 20 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devEL.LH[1].b, EL->LH[1].b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devEL.LH[1].Y, EL->LH[1].Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");

    hipMemcpy(devEL.LL.w, EL->LL.w, 20 * 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");
    hipMemcpy(devEL.LL.b, EL->LL.b, 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");
    hipMemcpy(devEL.LL.Y, EL->LL.Y, 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");


    //Y_ref is filled with output of io.x and weights and bias
    for (int i = 0; i < 5; i++)
    {

        Comp_Minimap(devEL,i);

        hipMemcpy(EL->io.Y_ref, devEL.LL.Y, 10 * sizeof(float), hipMemcpyDeviceToHost);
        cudaCheckErrors("cpy failed");

        real_cost += EL->SqrMean(i);
    }
    real_cost /= 5;

    printf("real_cost is %f \n", real_cost);
    

    calc_dw_db(EL, devEL, EL->LL.w, devEL.LL.w, EL->LL.dw, real_cost, 20 * 10);
    calc_dw_db(EL, devEL, EL->LL.b, devEL.LL.b,EL->LL.db,real_cost, 10);
    printf(" LL completed \n");
    calc_dw_db(EL, devEL, EL->LH[0].w, devEL.LH[0].w, EL->LH[0].dw, real_cost, 20 * 20);
    calc_dw_db(EL, devEL, EL->LH[0].b, devEL.LH[0].b, EL->LH[0].db, real_cost, 20);
    printf(" LH[0] completed \n");
    calc_dw_db(EL, devEL, EL->LH[1].w, devEL.LH[1].w, EL->LH[1].dw, real_cost, 20 * 20);
    calc_dw_db(EL, devEL, EL->LH[1].b, devEL.LH[1].b, EL->LH[1].db, real_cost, 20);
    printf(" LH[1] completed \n");
    calc_dw_db(EL, devEL, EL->LF.w, devEL.LF.w, EL->LF.dw, real_cost, 784 * 20);
    calc_dw_db(EL, devEL, EL->LF.b, devEL.LF.b, EL->LF.db, real_cost, 20);
    printf(" LF completed \n");



    hipMemcpy(EL->LH[0].Y, devEL.LH[0].Y, 20 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");

    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(EL->LF.Y, devEL.LF.Y, 20 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");

    hipMemcpy(EL->io.Y, devEL.LL.Y, 10 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");



    hipFree(devEL.LF.X);
    hipFree(devEL.LF.b);
    //hipFree(&devLF.db);
    hipFree(devEL.LF.w);
    hipFree(devEL.LF.refw);
    //hipFree(&devLF.dw);
    hipFree(devEL.LF.Y);

    hipFree(devEL.LH[0].b);
    //hipFree(&devLH[0].db);
    hipFree(devEL.LH[0].w);
    hipFree(devEL.LH[0].refw);
    //hipFree(&devLH[0].dw);
    hipFree(devEL.LH[0].Y);

    hipFree(devEL.LH[1].b);
    //hipFree(&devLH[1].db);
    hipFree(devEL.LH[1].w);
    hipFree(devEL.LH[1].refw);
    //hipFree(&devLH[1].dw);
    hipFree(devEL.LH[1].Y);

    hipFree(devEL.LL.b);
    //hipFree(&devLL.db);
    hipFree(devEL.LL.w);
    hipFree(devEL.LL.refw);
    //hipFree(&devLL.dw);
    hipFree(devEL.LL.Y);
    cudaCheckErrors("free mem failed");

    printf("-- cuda completed --\n");



    /*-------end last layer output coding-------*/






    //const int arraySize = 5;
    //const int a[arraySize] = { 1, 2, 3, 4, 5 };
    //const int b[arraySize] = { 10, 20, 30, 40, 50 };
    //int c[arraySize] = { 0 };

    //// Add vectors in parallel.
    //hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    return 1;
    //}

    //printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
    //    c[0], c[1], c[2], c[3], c[4]);

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    return 0;
}



void calc_dw_db(E_Layer* EL, E_Layer& devEL, float* ds,float* devds,float* dw, float real_cost,int arrsize)
{
    float calc_cost;
    for (int j = 0; j < arrsize; j++)
    {
        //EL->LL.w[j] += 0.1f;
        ds[j] += 0.001f;
        //hipMemcpy(&devEL.LL.w[j], &EL->LL.w[j], sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(&devds[j], &ds[j], sizeof(float), hipMemcpyHostToDevice);


        calc_cost = 0;

        for (int i = 0; i < 5; i++)
        {

            Comp_Minimap(devEL, i);

            hipMemcpy(EL->io.Y_ref, devEL.LL.Y, 10 * sizeof(float), hipMemcpyDeviceToHost);
            cudaCheckErrors("cpy failed");

            calc_cost += EL->SqrMean(i);

        }
        calc_cost /= 5;

        if (arrsize > 784)
        {
            //printf("cost %.4f,%4f,, ",calc_cost, 10 * (real_cost - calc_cost));

            dw[j] = 0.1f * (real_cost - calc_cost)/0.001f;
        }
        else
        {
            dw[j] = 1.f*(real_cost - calc_cost)/0.001f;
        }


        ds[j] -= 0.001f;
        hipMemcpy(&devds[j], &ds[j], sizeof(float), hipMemcpyHostToDevice);
    }
    for (int i = 0; i < arrsize; i++)
    {
        ds[i] += (5*dw[i]);
    }
}

int Comp_Minimap(E_Layer& devEL, int i)
{
    Sqs_X << <1, 784 >> > (&devEL.LF.X[784 * i]);

    //compute io --> layer first out(devLF.Y)
    Dot_w_X << <20, 784 >> > (devEL.LF.refw, devEL.LF.w, &devEL.LF.X[784 * i], 20, 784);
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LF");

    Comp_y << <1, 20 >> > (devEL.LF.Y, devEL.LF.refw, devEL.LF.b, 20, 784);
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LF");



    //compute devLF.Y --> devLH[0].Y
    Dot_w_X << <20, 20 >> > (devEL.LH[0].refw, devEL.LH[0].w, devEL.LF.Y, 20, 20);
    //cudaCheckErrors("fun failed LH[0]");
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LH[0]");

    Comp_y << <1, 20 >> > (devEL.LH[0].Y, devEL.LH[0].refw, devEL.LH[0].b, 20, 20);
   // cudaCheckErrors("fun failed LH[0]");
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LH[0]");


    //compute devLH[0].Y --> devLH[1].Y
    Dot_w_X << <20, 20 >> > (devEL.LH[1].refw, devEL.LH[1].w, devEL.LH[0].Y, 20, 20);
    //cudaCheckErrors("fun failed LH[1]");
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LH[1]");

    Comp_y << <1, 20 >> > (devEL.LH[1].Y, devEL.LH[1].refw, devEL.LH[1].b, 20, 20);
    //cudaCheckErrors("fun failed LH[1]");
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LH[1]");





    //compute devLH[1].Y --> devLL.Y
    Dot_w_X << <10, 20 >> > (devEL.LL.refw, devEL.LL.w, devEL.LH[1].Y, 10, 20);
    //cudaCheckErrors("fun failed LL");
    hipDeviceSynchronize();
    //cudaCheckErrors("synchronize failed LL");

    Comp_y << <1, 10 >> > (devEL.LL.Y, devEL.LL.refw, devEL.LL.b, 10, 20);
    //cudaCheckErrors("fun failed[10]");
    hipDeviceSynchronize();
    cudaCheckErrors("synchronize failed[10]");
}







int check(E_Layer* EL)
{
    Layer_f devLF;
    Layer_h devLH[2];
    Layer_l devLL;

    /*-----memeory allocation ---------*/
    hipMalloc((void**)&devLF.X, 784 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devLF.w, 784 * 20 * sizeof(float));
    hipMalloc((void**)&devLF.refw, 784 * 20 * sizeof(float));
    hipMalloc((void**)&devLF.b, 20 * sizeof(float));
    hipMalloc((void**)&devLF.Y, 20 * sizeof(float));

    // devHF[0] mem allocation
    //hipMalloc((void**)&devLF.X, 784 * sizeof(float));
    hipMalloc((void**)&devLH[0].w, 20 * 20 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devLH[0].refw, 20 * 20 * sizeof(float));
    hipMalloc((void**)&devLH[0].b, 20 * sizeof(float));
    hipMalloc((void**)&devLH[0].Y, 20 * sizeof(float));

    // devHF[0] mem allocation
    //hipMalloc((void**)&devLF.X, 784 * sizeof(float));
    hipMalloc((void**)&devLH[1].w, 20 * 20 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devLH[1].refw, 20 * 20 * sizeof(float));
    hipMalloc((void**)&devLH[1].b, 20 * sizeof(float));
    hipMalloc((void**)&devLH[1].Y, 20 * sizeof(float));

    //devLL mem allocation
    hipMalloc((void**)&devLL.w, 20 * 10 * sizeof(float));
    cudaCheckErrors("Elayer allocation error");
    hipMalloc((void**)&devLL.refw, 20 * 10 * sizeof(float));
    hipMalloc((void**)&devLL.b, 10 * sizeof(float));
    hipMalloc((void**)&devLL.Y, 10 * sizeof(float));


    hipMemcpy(devLF.w, EL->LF.w, 784 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");
    hipMemcpy(devLF.b, EL->LF.b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");
    hipMemcpy(devLF.Y, EL->LF.Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LF");

    hipMemcpy(devLH[0].w, EL->LH[0].w, 20 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devLH[0].b, EL->LH[0].b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devLH[0].Y, EL->LH[0].Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");

    hipMemcpy(devLH[1].w, EL->LH[1].w, 20 * 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devLH[1].b, EL->LH[1].b, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");
    hipMemcpy(devLH[1].Y, EL->LH[1].Y, 20 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LH");

    hipMemcpy(devLL.w, EL->LL.w, 20 * 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");
    hipMemcpy(devLL.b, EL->LL.b, 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");
    hipMemcpy(devLL.Y, EL->LL.Y, 10 * sizeof(float), hipMemcpyHostToDevice);
    cudaCheckErrors("copy failed LL");



    /*--------mini map loop ---------*/

        hipMemcpy(devLF.X, EL->io.User_X, 784 * sizeof(float), hipMemcpyHostToDevice);
        cudaCheckErrors("copy failed from io failed");



        /*-------upto last layer output coding-------*/
        Sqs_X << <1, 784 >> > (devLF.X);
        hipMemcpy(EL->LF.Y, devLF.X, 20 * sizeof(float), hipMemcpyDeviceToHost);


        //compute io --> layer first out(devLF.Y)
        Dot_w_X << <20, 784 >> > (devLF.refw, devLF.w, devLF.X, 20, 784);
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LF");

        Comp_y << <1, 20 >> > (devLF.Y, devLF.refw, devLF.b, 20, 784);
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LF");





        //compute devLF.Y --> devLH[0].Y
        Dot_w_X << <20, 20 >> > (devLH[0].refw, devLH[0].w, devLF.Y, 20, 20);
        cudaCheckErrors("fun failed LH[0]");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LH[0]");

        Comp_y << <1, 20 >> > (devLH[0].Y, devLH[0].refw, devLH[0].b, 20, 20);
        cudaCheckErrors("fun failed LH[0]");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LH[0]");


        //compute devLH[0].Y --> devLH[1].Y
        Dot_w_X << <20, 20 >> > (devLH[1].refw, devLH[1].w, devLH[0].Y, 20, 20);
        cudaCheckErrors("fun failed LH[1]");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LH[1]");

        Comp_y << <1, 20 >> > (devLH[1].Y, devLH[1].refw, devLH[1].b, 20, 20);
        cudaCheckErrors("fun failed LH[1]");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LH[1]");





        //compute devLH[1].Y --> devLL.Y
        Dot_w_X << <10, 20 >> > (devLL.refw, devLL.w, devLH[1].Y, 10, 20);
        cudaCheckErrors("fun failed LL");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed LL");

        Comp_y << <1, 10 >> > (devLL.Y, devLL.refw, devLL.b, 10, 20);
        cudaCheckErrors("fun failed[10]");
        hipDeviceSynchronize();
        cudaCheckErrors("synchronize failed[10]");


    
    //cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(EL->LF.Y, devLF.Y, 20 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");

    hipMemcpy(EL->LH[0].Y, devLH[0].Y, 20 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");

    hipMemcpy(EL->LH[1].Y, devLH[1].Y, 20 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");

    hipMemcpy(EL->LL.Y, devLL.Y, 10 * sizeof(float), hipMemcpyDeviceToHost);
    cudaCheckErrors("cpy failed");



    hipFree(devLF.X);
    hipFree(devLF.b);
    //hipFree(&devLF.db);
    hipFree(devLF.w);
    hipFree(devLF.refw);
    //hipFree(&devLF.dw);
    hipFree(devLF.Y);

    hipFree(devLH[0].b);
    //hipFree(&devLH[0].db);
    hipFree(devLH[0].w);
    hipFree(devLH[0].refw);
    //hipFree(&devLH[0].dw);
    hipFree(devLH[0].Y);

    hipFree(devLH[1].b);
    //hipFree(&devLH[1].db);
    hipFree(devLH[1].w);
    hipFree(devLH[1].refw);
    //hipFree(&devLH[1].dw);
    hipFree(devLH[1].Y);

    hipFree(devLL.b);
    //hipFree(&devLL.db);
    hipFree(devLL.w);
    hipFree(devLL.refw);
    //hipFree(&devLL.dw);
    hipFree(devLL.Y);
    cudaCheckErrors("free mem failed");

    printf("-- cuda completed for check --\n");



    /*-------end last layer output coding-------*/


    return 0;
}


// Helper function for using CUDA to add vectors in parallel.
//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
//{
//    int *dev_a = 0;
//    int *dev_b = 0;
//    int *dev_c = 0;
//    hipError_t cudaStatus;
//
//    // Choose which GPU to run on, change this on a multi-GPU system.
//    cudaStatus = hipSetDevice(0);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//        goto Error;
//    }
//
//    // Allocate GPU buffers for three vectors (two input, one output)    .
//    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMalloc failed!");
//        goto Error;
//    }
//
//    // Copy input vectors from host memory to GPU buffers.
//    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//    // Launch a kernel on the GPU with one thread for each element.
//    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);
//
//    // Check for any errors launching the kernel
//    cudaStatus = hipGetLastError();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//        goto Error;
//    }
//    
//    // hipDeviceSynchronize waits for the kernel to finish, and returns
//    // any errors encountered during the launch.
//    cudaStatus = hipDeviceSynchronize();
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//        goto Error;
//    }
//
//    // Copy output vector from GPU buffer to host memory.
//    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
//    if (cudaStatus != hipSuccess) {
//        fprintf(stderr, "hipMemcpy failed!");
//        goto Error;
//    }
//
//Error:
//    hipFree(dev_c);
//    hipFree(dev_a);
//    hipFree(dev_b);
//    
//    return cudaStatus;
//}
