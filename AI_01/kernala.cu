
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include "Neuron.h"
//#include "kernel.cuh"
//
//
//#define cudaCheckErrors(msg) \
//    do { \
//        cudaError_t __err = cudaGetLastError(); \
//        if (__err != cudaSuccess) { \
//            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
//                msg, cudaGetErrorString(__err), \
//                __FILE__, __LINE__); \
//            fprintf(stderr, "*** FAILED - ABORTING\n"); \
//            exit(1); \
//        } \
//    } while (0)
//
////cudaCheckErrors("Elayer allocation error");
//__global__ void Sqs_X(float* X)
//{
//    if (threadIdx.x < 784)
//    {
//        X[threadIdx.x] = (1.0f / (1.0f + powf(2.71828f, -(X[threadIdx.x]))));
//
//    }
//}
//__global__ void Dot_w_X(float* refw, float* w, float* X, int w_row, int w_col)
//{
//    if (blockIdx.x < w_row && threadIdx.x < w_col)
//    {
//        refw[blockIdx.x * w_col + threadIdx.x] = (w[blockIdx.x * w_col + threadIdx.x] * X[threadIdx.x]);
//
//    }
//}
//
//
//int Kernal_alloc_mem(E_Layer& devEL)
//{
//
//    /*-----memeory allocation ---------*/
//    cudaMalloc((void**)&devEL.LF.X, 784 * 5 * sizeof(float));
//    cudaCheckErrors("Elayer allocation error");
//    cudaMalloc((void**)&devEL.LF.w, 784 * 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LF.refw, 784 * 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LF.b, 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LF.Y, 20 * sizeof(float));
//
//    // devHF[0] mem allocation
//    //cudaMalloc((void**)&devLF.X, 784 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[0].w, 20 * 20 * sizeof(float));
//    cudaCheckErrors("Elayer allocation error");
//    cudaMalloc((void**)&devEL.LH[0].refw, 20 * 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[0].b, 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[0].Y, 20 * sizeof(float));
//
//    // devHF[0] mem allocation
//    //cudaMalloc((void**)&devLF.X, 784 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[1].w, 20 * 20 * sizeof(float));
//    cudaCheckErrors("Elayer allocation error");
//    cudaMalloc((void**)&devEL.LH[1].refw, 20 * 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[1].b, 20 * sizeof(float));
//    cudaMalloc((void**)&devEL.LH[1].Y, 20 * sizeof(float));
//
//    //devLL mem allocation
//    cudaMalloc((void**)&devEL.LL.w, 20 * 10 * sizeof(float));
//    cudaCheckErrors("Elayer allocation error");
//    cudaMalloc((void**)&devEL.LL.refw, 20 * 10 * sizeof(float));
//    cudaMalloc((void**)&devEL.LL.b, 10 * sizeof(float));
//    cudaMalloc((void**)&devEL.LL.Y, 10 * sizeof(float));
//
//}
//
//
//int Kernal_free_mem(E_Layer& devEL)
//{
//    cudaFree(devEL.LF.X);
//    cudaFree(devEL.LF.b);
//    //cudaFree(&devLF.db);
//    cudaFree(devEL.LF.w);
//    cudaFree(devEL.LF.refw);
//    //cudaFree(&devLF.dw);
//    cudaFree(devEL.LF.Y);
//
//    cudaFree(devEL.LH[0].b);
//    //cudaFree(&devLH[0].db);
//    cudaFree(devEL.LH[0].w);
//    cudaFree(devEL.LH[0].refw);
//    //cudaFree(&devLH[0].dw);
//    cudaFree(devEL.LH[0].Y);
//
//    cudaFree(devEL.LH[1].b);
//    //cudaFree(&devLH[1].db);
//    cudaFree(devEL.LH[1].w);
//    cudaFree(devEL.LH[1].refw);
//    //cudaFree(&devLH[1].dw);
//    cudaFree(devEL.LH[1].Y);
//
//    cudaFree(devEL.LL.b);
//    //cudaFree(&devLL.db);
//    cudaFree(devEL.LL.w);
//    cudaFree(devEL.LL.refw);
//    //cudaFree(&devLL.dw);
//    cudaFree(devEL.LL.Y);
//    cudaCheckErrors("free mem failed");
//
//    printf("-- cuda freed memory --\n");
//}
//
//
//
//int Kernal_cpy_host(E_Layer& devEL, E_Layer& EL)
//{
//    cudaMemcpy(devEL.LF.X, EL.io.X, 784 * 5 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed from io failed");
//
//    cudaMemcpy(devEL.LF.w, EL.LF.w, 784 * 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LF");
//    cudaMemcpy(devEL.LF.b, EL.LF.b, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LF");
//    cudaMemcpy(devEL.LF.Y, EL.LF.Y, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LF");
//
//    cudaMemcpy(devEL.LH[0].w, EL.LH[0].w, 20 * 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//    cudaMemcpy(devEL.LH[0].b, EL.LH[0].b, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//    cudaMemcpy(devEL.LH[0].Y, EL.LH[0].Y, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//
//    cudaMemcpy(devEL.LH[1].w, EL.LH[1].w, 20 * 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//    cudaMemcpy(devEL.LH[1].b, EL.LH[1].b, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//    cudaMemcpy(devEL.LH[1].Y, EL.LH[1].Y, 20 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LH");
//
//    cudaMemcpy(devEL.LL.w, EL.LL.w, 20 * 10 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LL");
//    cudaMemcpy(devEL.LL.b, EL.LL.b, 10 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LL");
//    cudaMemcpy(devEL.LL.Y, EL.LL.Y, 10 * sizeof(float), cudaMemcpyHostToDevice);
//    cudaCheckErrors("copy failed LL");
//
//
//}
//
//int Kernal_cpy_device(float* des, float* src, int size)
//{
//    //cudaMemcpy()
//}
